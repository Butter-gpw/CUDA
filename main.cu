#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <sys/time.h>
#include "gemm.cuh"
#include "utils.cuh"
#include "reduce.cuh"
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])


__global__ void warp_reduce_sum_test(float* output, float *input, int n){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n){
        float val = input[idx];
        val = warp_reduce_sum(val);
        if(threadIdx.x % WARP_SIZE==0){
            output[idx / WARP_SIZE]=val;
            printf("warpID:%d, %f\n", blockIdx.x, val);
        }
    }
}

int main(void){
    setGPU(1);
    float *input, *output,*host;
    int n=1024;
    host = (float*)malloc(n * sizeof(float));
    for(int i=0;i<n;++i){
        host[i] = i;
    }
    float* hostSum, *deviceSum;
    hostSum = new float(0);
    hipMalloc(&input, n * sizeof(float));
    hipMalloc(&output, (n/ WARP_SIZE)* sizeof(float));
    hipMalloc(&deviceSum, sizeof(float ));
    hipMemcpy(deviceSum, hostSum,sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(input, host,n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 128;
    int numBlocks = (n + blockSize-1) /blockSize;
    block_all_reduce_sum_vec4<<<numBlocks, blockSize>>>(input, deviceSum, n);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    hipMemcpy(hostSum, deviceSum, sizeof(float ), hipMemcpyDeviceToHost);
    printf("sum:%f\n", *hostSum);
    hipFree(input);
    hipFree(output);
    hipFree(deviceSum);
    delete hostSum;

    return 0;
}