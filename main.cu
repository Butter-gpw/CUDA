#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <sys/time.h>
#include "gemm.cuh"
#include "utils.cuh"
#include "activate.cuh"
#include "softmax.cuh"
#include "norm.cuh"
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])


int main(void){
    setGPU(2);
    float *input, *output,*host;
    int n=1024;
    host = (float*)malloc(n * sizeof(float));
    for(int i=0;i<n;++i){
        host[i] = i;
    }
    float* hostSum, *deviceSum;
    hostSum = new float(0);
    hipMalloc(&input, n * sizeof(float));
    hipMalloc(&output, (n/ WARP_SIZE)* sizeof(float));
    hipMalloc(&deviceSum, sizeof(float ));
    hipMemcpy(deviceSum, hostSum,sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(input, host,n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 128;
    int numBlocks = (n + blockSize-1) /blockSize;
    block_all_reduce_sum_vec4<<<numBlocks, blockSize>>>(input, deviceSum, n);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    hipMemcpy(hostSum, deviceSum, sizeof(float ), hipMemcpyDeviceToHost);
    printf("sum:%f\n", *hostSum);
    hipFree(input);
    hipFree(output);
    hipFree(deviceSum);
    delete hostSum;

    return 0;
}